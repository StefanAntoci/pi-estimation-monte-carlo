#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>
#include <random>
#include <hiprand.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(hiprandState* state);
__global__ void monte_carlo_pi_kernel(hiprandState* state, int* count, int m);

__global__ void setup_kernel(hiprandState* state)
{
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	hiprand_init(123, index, 0, &state[index]);
}




__global__ void monte_carlo_pi_kernel(hiprandState* state, int* count, int m)
{
	unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;

	__shared__ int memory[256];
	memory[threadIdx.x] = 0;
	__syncthreads();


	unsigned int temp = 0;
	while (temp < m) {
		float x = hiprand_uniform(&state[index]);
		float y = hiprand_uniform(&state[index]);
		float r = x * x + y * y;

		if (r <= 1) {
			memory[threadIdx.x]++;
			
		}
		temp++;
	}
	__syncthreads();
	// reduction
	int i = blockDim.x / 2;
	while (i != 0) {
		if (threadIdx.x < i) {
			memory[threadIdx.x] += memory[threadIdx.x + i];
		}

		i /= 2;
		__syncthreads();
	}


	// update to our global variable count
	if (threadIdx.x == 0) {
		atomicAdd(count, memory[0]);
	}
}





int main()
{
	unsigned int n = 256* 256;
	unsigned int m = 1000;
	int* h_count;
	int* d_count;
	hiprandState* d_state;
	float pi;


	// allocate memory
	h_count = (int*)malloc(n * sizeof(int));
	hipMalloc((void**)&d_count, n * sizeof(int));
	hipMalloc((void**)&d_state, n * sizeof(hiprandState));
	hipMemset(d_count, 0, sizeof(int));


	// set up timing stuff
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start, 0);


	// set kernel
	dim3 gridSize = 256;
	dim3 blockSize = 256;
	setup_kernel << < gridSize, blockSize >> > (d_state);


	// monti carlo kernel
	monte_carlo_pi_kernel << <gridSize, blockSize >> > (d_state, d_count, m);


	// copy results back to the host
	hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);


	// display results and timings for gpu
	pi = *h_count * 4.0 / (n * m);
	std::cout << "Approximate pi calculated on GPU is: " << pi << " and calculation took " << gpu_elapsed_time << std::endl;

	// delete memory
	free(h_count);
	hipFree(d_count);
	hipFree(d_state);
}
